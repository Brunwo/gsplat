#include "hip/hip_runtime.h"
#include "backward.cuh"
#include "forward.cuh"
#include "helpers.cuh"
#include "rasterize.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>

#define CHECK_CUDA(x)                                                          \
    AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)                                                    \
    AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)                                                         \
    CHECK_CUDA(x);                                                             \
    CHECK_CONTIGUOUS(x)

std::
    tuple<
        torch::Tensor, // dL_dxy
        torch::Tensor, // dL_dconic
        torch::Tensor, // dL_dcolors
        torch::Tensor  // dL_dopacity
        >
    nd_rasterize_backward_tensor(
        const unsigned img_height,
        const unsigned img_width,
        const torch::Tensor &gaussians_ids_sorted,
        const torch::Tensor &tile_bins,
        const torch::Tensor &xys,
        const torch::Tensor &conics,
        const torch::Tensor &colors,
        const torch::Tensor &opacities,
        const torch::Tensor &background,
        const torch::Tensor &final_Ts,
        const torch::Tensor &final_idx,
        const torch::Tensor &v_output // dL_dout_color
    ) {

    CHECK_INPUT(xys);
    CHECK_INPUT(colors);

    if (xys.ndimension() != 2 || xys.size(1) != 2) {
        AT_ERROR("xys must have dimensions (num_points, 2)");
    }

    if (colors.ndimension() != 2) {
        AT_ERROR("colors must have 2 dimensions");
    }

    const int num_points = xys.size(0);
    const dim3 tile_bounds = {
        (img_width + BLOCK_X - 1) / BLOCK_X,
        (img_height + BLOCK_Y - 1) / BLOCK_Y,
        1
    };
    const dim3 block(BLOCK_X, BLOCK_Y, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    torch::Tensor v_xy = torch::zeros({num_points, 2}, xys.options());
    torch::Tensor v_conic = torch::zeros({num_points, 3}, xys.options());
    torch::Tensor v_colors =
        torch::zeros({num_points, channels}, xys.options());
    torch::Tensor v_opacity = torch::zeros({num_points, 1}, xys.options());

    torch::Tensor workspace;
    if (channels > 3) {
        workspace = torch::zeros(
            {img_height, img_width, channels},
            xys.options().dtype(torch::kFloat32)
        );
    } else {
        workspace = torch::zeros({0}, xys.options().dtype(torch::kFloat32));
    }

    nd_rasterize_backward_impl(
        tile_bounds,
        block,
        img_size,
        channels,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float2 *)xys.contiguous().data_ptr<float>(),
        (float3 *)conics.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        background.contiguous().data_ptr<float>(),
        final_Ts.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        v_output.contiguous().data_ptr<float>(),
        (float2 *)v_xy.contiguous().data_ptr<float>(),
        (float3 *)v_conic.contiguous().data_ptr<float>(),
        v_colors.contiguous().data_ptr<float>(),
        v_opacity.contiguous().data_ptr<float>(),
        workspace.data_ptr<float>()
    );

    return std::make_tuple(v_xy, v_conic, v_colors, v_opacity);
}

std::
    tuple<
        torch::Tensor, // dL_dxy
        torch::Tensor, // dL_dconic
        torch::Tensor, // dL_dcolors
        torch::Tensor  // dL_dopacity
        >
    rasterize_backward_tensor(
        const unsigned img_height,
        const unsigned img_width,
        const torch::Tensor &gaussians_ids_sorted,
        const torch::Tensor &tile_bins,
        const torch::Tensor &xys,
        const torch::Tensor &conics,
        const torch::Tensor &colors,
        const torch::Tensor &opacities,
        const torch::Tensor &background,
        const torch::Tensor &final_Ts,
        const torch::Tensor &final_idx,
        const torch::Tensor &v_output // dL_dout_color
    ) {

    CHECK_INPUT(xys);
    CHECK_INPUT(colors);

    if (xys.ndimension() != 2 || xys.size(1) != 2) {
        AT_ERROR("xys must have dimensions (num_points, 2)");
    }

    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }

    const int num_points = xys.size(0);
    const dim3 tile_bounds = {
        (img_width + BLOCK_X - 1) / BLOCK_X,
        (img_height + BLOCK_Y - 1) / BLOCK_Y,
        1
    };
    const dim3 block(BLOCK_X, BLOCK_Y, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    torch::Tensor v_xy = torch::zeros({num_points, 2}, xys.options());
    torch::Tensor v_conic = torch::zeros({num_points, 3}, xys.options());
    torch::Tensor v_colors =
        torch::zeros({num_points, channels}, xys.options());
    torch::Tensor v_opacity = torch::zeros({num_points, 1}, xys.options());

    rasterize_backward_impl(
        tile_bounds,
        block,
        img_size,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float2 *)xys.contiguous().data_ptr<float>(),
        (float3 *)conics.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        final_Ts.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        (float2 *)v_xy.contiguous().data_ptr<float>(),
        (float3 *)v_conic.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacity.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_xy, v_conic, v_colors, v_opacity);
}
